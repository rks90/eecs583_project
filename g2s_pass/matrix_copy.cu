
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>

using namespace std;


//Define TILE_DIM and BLOCK_ROWS
const int TILE_DIM = 32;
const int BLOCK_ROWS = 8;

__global__ void copy(float *odata, const float *idata)
{
  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
    odata[(y+j)*width + x] = idata[(y+j)*width + x];
}

int main(void)
{
	//Multiple of 32
	//int N = 1<<20; //1M elements
	int N = 100*32; 
	
	//Allocate Unified Memory -- accessible from CPU or GPU
	float *x, *y;
	hipMallocManaged(&x, N*N*sizeof(float));
	hipMallocManaged(&y, N*N*sizeof(float));
	
	//initialize x
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			x[i*N + j] = 1.0f *(i*N + j) ;
		}
	}

	ofstream outfile;
	outfile.open("copy_out.txt");

	outfile << "Input Matrix:" << endl;
  	// Output X Matrix
  	for (int i = 0; i < N; i++) {
  		for (int j = 0; j < N; j++) {
			outfile << x[i*N+j] << " ";
		}
		outfile << "\n";
	}
	dim3 numThreads(TILE_DIM,BLOCK_ROWS,1);
	dim3 numBlocks(N/(TILE_DIM),N/(TILE_DIM),1);
	printf("numBlocks: %d %d %d. numThreads: %d %d %d\n",numBlocks.x, numBlocks.y, numBlocks.z, numThreads.x, numThreads.y, numThreads.z);
	// Run kernel on 1M elements on the CPU
  	copy<<<numBlocks, numThreads>>>(y, x);

	//Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();


	outfile << "Output Matrix:" << endl;
  	// Output Matrix
  	for (int i = 0; i < N; i++) {
  		for (int j = 0; j < N; j++) {
			outfile << y[i*N+j] << " ";
		}
		outfile << "\n";
	}

  	// Free memory
	hipFree(x);
	hipFree(y);

	outfile.close();

  	return 0;
}
