
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

//function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
	for (int i = 0; i < n; i++)
	y[i] = x[i] + y[i];
}

int main(void)
{
	//int N = 1<<20; //1M elements
	int N = 100; //100 elements
	
	std::cout << "DEBUG0:" << std::endl;
	//Allocate Unified Memory -- accessible from CPU or GPU
	float *x, *y;
	hipMallocManaged(&x, N*sizeof(float));
	hipMallocManaged(&x, N*sizeof(float));
	
	//initialize x and y arrays on the host
	for (int i = 0; i < N; i++) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}
	std::cout << "DEBUG1:" << std::endl;
	// Run kernel on 1M elements on the CPU
  	add<<<1,1>>>(N, x, y);
	std::cout << "DEBUG2:" << std::endl;

	//Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();

	std::cout << "DEBUG3:" << std::endl;
  	// Check for errors (all values should be 3.0f)
  	float maxError = 0.0f;
  	for (int i = 0; i < N; i++)
    		maxError = fmax(maxError, fabs(y[i]-3.0f));
  	std::cout << "Max error: " << maxError << std::endl;
	std::cout << "DEBUG4:" << std::endl;

  	// Free memory
	hipFree(x);
	hipFree(y);
	std::cout << "DEBUG5:" << std::endl;

  	return 0;
}
