
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

//function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
	int index = threadIdx.x;
	int stride = blockDim.x;
	for (int i = index; i < n; i+= stride)
		y[i] = x[i] + y[i];
}

int main(void)
{
	int N = 1<<20; //1M elements
	//int N = 100; //100 elements

	int blockSize = 256;
	//int numBlocks = (N+blockSize -1) / blocksize;
	int numBlocks = 1;
	
	//Allocate Unified Memory -- accessible from CPU or GPU
	float *x, *y, *d_x, *d_y;
	x = (float *)malloc(N*sizeof(float));
	y = (float *)malloc(N*sizeof(float));
	hipMalloc(&d_x, N*sizeof(float));
	hipMalloc(&d_y, N*sizeof(float));
	
	//initialize x and y arrays on the host
	for (int i = 0; i < N; i++) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}

	//Copy
	hipMemcpy(d_x,x, N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_y,y, N*sizeof(float), hipMemcpyHostToDevice);

	// Run kernel on 1M elements on the CPU
  	add<<<numBlocks,blockSize>>>(N, d_x, d_y);

	//Wait for GPU to finish before accessing on host
	//cudaDeviceSynchronize();

	hipMemcpy(y,d_y, N*sizeof(float), hipMemcpyDeviceToHost);

  	// Check for errors (all values should be 3.0f)
  	float maxError = 0.0f;
  	for (int i = 0; i < N; i++)
    		maxError = fmax(maxError, fabs(y[i]-3.0f));
  	std::cout << "Max error: " << maxError << std::endl;

  	// Free memory
	hipFree(d_x);
	hipFree(d_y);
	free(x);
	free(y);

  	return 0;
}
