
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include<hip/hip_runtime_api.h>

//function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
	for (int i = 0; i < n; i++)
	y[i] = x[i] + y[i];
}

int main(void)
{
	//cudaProfilerStart();
	int N = 1<<20; //1M elements
	//int N = 100; //100 elements
	
	//Allocate Unified Memory -- accessible from CPU or GPU
	float *x, *y, *d_x, *d_y;
	x = (float *)malloc(N*sizeof(float));
	y = (float *)malloc(N*sizeof(float));
	hipMalloc(&d_x, N*sizeof(float));
	hipMalloc(&d_y, N*sizeof(float));
	
	//initialize x and y arrays on the host
	for (int i = 0; i < N; i++) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}

	//Copy
	hipMemcpy(d_x,x, N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_y,y, N*sizeof(float), hipMemcpyHostToDevice);

	// Run kernel on 1M elements on the CPU
  	add<<<1,1>>>(N, d_x, d_y);

	//Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();

	hipMemcpy(y,d_y, N*sizeof(float), hipMemcpyDeviceToHost);

  	// Check for errors (all values should be 3.0f)
  	float maxError = 0.0f;
  	for (int i = 0; i < N; i++)
    		maxError = fmax(maxError, fabs(y[i]-3.0f));
  	std::cout << "Max error: " << maxError << std::endl;

  	// Free memory
	hipFree(d_x);
	hipFree(d_y);
	free(x);
	free(y);
	//cudaProfilerStop();
  	return 0;
}
