
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <stdio.h>

//function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
	int index = blockIdx.x* blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	/*printf("threadIdx.x = %d threadIdx.y = %d threadIdx.z = %d\
		blockIdx.x = %d blockIdx.y = %d blockIdx.z = %d\
		blockDim.x = %d blockDim.y = %d blockDim.z = %d\
		gridDim.x = %d gridDim.y = %d gridDim.z = %d\n",\
		threadIdx.x, threadIdx.y, threadIdx.z,\
		blockIdx.x,blockIdx.y,blockIdx.z,\
		blockDim.x,blockDim.y,blockDim.z,\
		gridDim.x,gridDim.y,gridDim.z);*/
	
	
	for (int i = index; i < n; i+= stride)
		y[i] = x[i] + y[i];
}

int main(void)
{
	int N = 1<<20; //1M elements
	//int N = 100; //100 elements

	int blockSize = 256;
	int numBlocks = (N+blockSize -1) / blockSize;
	//int numBlocks = 1;
	
	//Allocate Unified Memory -- accessible from CPU or GPU
	float *x, *y;
	hipMallocManaged(&x, N*sizeof(float));
	hipMallocManaged(&y, N*sizeof(float));
	
	//initialize x and y arrays on the host
	for (int i = 0; i < N; i++) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}

	// Run kernel on 1M elements on the CPU
  	add<<<numBlocks,blockSize>>>(N, x, y);

	//Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();

  	// Check for errors (all values should be 3.0f)
  	float maxError = 0.0f;
  	for (int i = 0; i < N; i++)
    		maxError = fmax(maxError, fabs(y[i]-3.0f));
  	std::cout << "Max error: " << maxError << std::endl;

  	// Free memory
	hipFree(x);
	hipFree(y);

  	return 0;
}
