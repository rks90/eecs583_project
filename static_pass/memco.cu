
#include <hip/hip_runtime.h>
#include <iostream>

__device__ int memr[2048];
__device__ int memg[2048];
__device__ int memb[2048];
__shared__ int mem_out[2048];

__global__ void func() {
	int i = blockIdx.x * 256 + threadIdx.x;
	// This line is different from the line in our presentation,
	// but it works similarly and yields similar results.
	mem_out[i] = memr[(i*4 + 1) % 2048] * memr[i*4 % 2048]
			   + memg[i*4 % 2048] + memb[i*4 % 2048];
}

int main() {
	for (int i = 0; i < 1000000; i++) {
		func<<<8, 256>>>();
		hipDeviceSynchronize();
	}
	return 0;
}

